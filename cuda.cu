
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>

typedef struct {
    int user_id;
    int item_id;
    float rating;
} Data;

// Function to load CSV
Data* load_csv(const char* path, int* num_rows, char* header) {
    FILE* file = fopen(path, "r");
    if (!file) {
        fprintf(stderr, "Error opening file: %s\n", path);
        exit(EXIT_FAILURE);
    }

    printf("File opened\n");

    int capacity = 926055; // Hardcoded number of lines, adjust if necessary
    Data* data = (Data*)malloc(capacity * sizeof(Data));
    if (!data) {
        fprintf(stderr, "Memory allocation failed\n");
        fclose(file);
        exit(EXIT_FAILURE);
    }
    *num_rows = 0;

    printf("Memory allocated\n");

    // Remove the first line (header)
    if (fgets(header, 256, file) == NULL) {
        fprintf(stderr, "Error reading header line\n");
        free(data);
        fclose(file);
        exit(EXIT_FAILURE);
    }
    printf("Header read: %s\n", header);

    while (fscanf(file, "%d,%d,%f\n", &data[*num_rows].user_id, &data[*num_rows].item_id, &data[*num_rows].rating) != EOF) {
        (*num_rows)++;
        if (*num_rows >= capacity) {
            fprintf(stderr, "Exceeded pre-allocated capacity\n");
            break;
        }
        if (*num_rows <= 10) { // Print the first few rows for debugging
            printf("Row %d loaded: user_id=%d, item_id=%d, rating=%.2f\n", *num_rows, data[*num_rows-1].user_id, data[*num_rows-1].item_id, data[*num_rows-1].rating);
        }
    }

    printf("Data loaded, total rows: %d\n", *num_rows);
    fclose(file);
    printf("File closed\n");

    return data;
}

// Function to generate labels
int* generate_labels(Data* data, int num_rows, int* num_labels, int use_user_id) {
    int* labels = (int*)malloc(num_rows * sizeof(int));
    int id = 0;

    for (int i = 0; i < num_rows; i++) {
        int value = use_user_id ? data[i].user_id : data[i].item_id;
        int found = 0;
        for (int j = 0; j < i; j++) {
            int compare_value = use_user_id ? data[j].user_id : data[j].item_id;
            if (value == compare_value) {
                labels[i] = labels[j];
                found = 1;
                break;
            }
        }
        if (!found) {
            labels[i] = id++;
        }
        if (i < 10) { // Print the first few labels for debugging
            printf("Row %d: value=%d, label=%d\n", i, value, labels[i]);
        }
    }

    *num_labels = id;
    return labels;
}

// Function to export a matrix to a file
void export_matrix_to_file(float* matrix, int rows, int cols, const char* filename) {
    FILE* file = fopen(filename, "w");
    if (file == NULL) {
        fprintf(stderr, "Error opening file: %s\n", filename);
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            fprintf(file, "%f ", matrix[i * cols + j]);
        }
        fprintf(file, "\n");
    }

    fclose(file);
    printf("Matrix exported to %s\n", filename);
}

// CUDA kernel to transpose a matrix
__global__ void transpose_kernel(float* X, float* XT, int rows, int cols) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < rows && j < cols) {
        XT[j * rows + i] = X[i * cols + j];
    }
}

// CUDA kernel to multiply two matrices
__global__ void matrix_multiply_kernel(float* A, float* B, float* C, int A_rows, int A_cols, int B_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < A_rows && col < B_cols) {
        float sum = 0.0;
        for (int k = 0; k < A_cols; k++) {
            sum += A[row * A_cols + k] * B[k * B_cols + col];
        }
        C[row * B_cols + col] = sum;
    }
}

// CUDA kernel to build the G matrix
__global__ void build_g_matrix_kernel(float* G, Data* data, int* user_labels, int num_rows, int num_user_labels, float reg) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_rows) {
        int user_label = user_labels[idx];
        atomicAdd(&G[user_label * num_user_labels + user_label], data[idx].rating * data[idx].rating);
    }
    if (idx < num_user_labels) {
        G[idx * num_user_labels + idx] += reg;
    }
}

// CUDA kernel to invert the G matrix
__global__ void invert_matrix_kernel(float* G, float* P, int num_labels) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_labels && j < num_labels) {
        P[i * num_labels + j] = (i == j) ? 1.0 / G[i * num_labels + i] : -G[i * num_labels + j] / (G[i * num_labels + i] * G[j * num_labels + j]);
    }
}

// CUDA kernel to build the B matrix
__global__ void build_b_matrix_kernel(float* B, float* P, int num_labels) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num_labels && j < num_labels) {
        B[i * num_labels + j] = P[i * num_labels + j] / (-P[i * num_labels + i]);
        if (i == j) {
            B[i * num_labels + i] += 1.0;
        }
    }
}

int main() {
    auto start = std::chrono::high_resolution_clock::now(); // Start time

    printf("Begin\n");
    const char* path = "/content/CAD-2024/goodbooksData/trainBooksSmall.csv";
    int num_rows;

    printf("Start\n");
    // Allocate space for the header
    char header[256];
    Data* data = load_csv(path, &num_rows, header);

    printf("Imported CSV\n");

    int num_user_labels, num_item_labels;
    int* item_labels = generate_labels(data, num_rows, &num_item_labels, 1);  // Switched to 1 for item_id
    int* user_labels = generate_labels(data, num_rows, &num_user_labels, 0);  // Switched to 0 for user_id

    printf("Generated labels\n");
    printf("num_user_labels: %d, num_item_labels: %d\n", num_user_labels, num_item_labels);

    float** G = (float**)malloc(num_user_labels * sizeof(float*));  // Based on number of users (switched)
    float** P = (float**)malloc(num_user_labels * sizeof(float*));  // Based on number of users (switched)
    float** B = (float**)malloc(num_user_labels * sizeof(float*));  // Based on number of users (switched)
    for (int i = 0; i < num_user_labels; i++) {
        G[i] = (float*)malloc(num_user_labels * sizeof(float));    // Based on number of users (switched)
        P[i] = (float*)malloc(num_user_labels * sizeof(float));    // Based on number of users (switched)
        B[i] = (float*)malloc(num_user_labels * sizeof(float));    // Based on number of users (switched)
    }

    printf("Allocated memory for matrices\n");
    printf("Matrix G dimensions: %d x %d\n", num_user_labels, num_user_labels);
    printf("Matrix P dimensions: %d x %d\n", num_user_labels, num_user_labels);
    printf("Matrix B dimensions: %d x %d\n", num_user_labels, num_user_labels);

    // Allocate device memory
    float* d_G, * d_P, * d_B, * d_X, * d_XT, * d_C;
    Data* d_data;
    int* d_user_labels, * d_item_labels;
    hipMalloc((void**)&d_G, num_user_labels * num_user_labels * sizeof(float));
    hipMalloc((void**)&d_P, num_user_labels * num_user_labels * sizeof(float));
    hipMalloc((void**)&d_B, num_user_labels * num_user_labels * sizeof(float));
    hipMalloc((void**)&d_X, num_user_labels * num_item_labels * sizeof(float));
    hipMalloc((void**)&d_XT, num_item_labels * num_user_labels * sizeof(float));
    hipMalloc((void**)&d_C, num_user_labels * num_user_labels * sizeof(float));
    hipMalloc((void**)&d_data, num_rows * sizeof(Data));
    hipMalloc((void**)&d_user_labels, num_rows * sizeof(int));
    hipMalloc((void**)&d_item_labels, num_rows * sizeof(int));

    // Copy data to device
    hipMemcpy(d_data, data, num_rows * sizeof(Data), hipMemcpyHostToDevice);
    hipMemcpy(d_user_labels, user_labels, num_rows * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_item_labels, item_labels, num_rows * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernels
    int blockSize = 256;
    int numBlocks = (num_rows + blockSize - 1) / blockSize;
    printf("num de blocos: %d\n", numBlocks);
    build_g_matrix_kernel<<<numBlocks, blockSize>>>(d_G, d_data, d_user_labels, num_rows, num_user_labels, 250.0);

    printf("Matrix X dimensions: %d x %d\n", num_item_labels, num_user_labels);
    printf("Matrix XT dimensions (before transpose): %d x %d\n", num_user_labels, num_item_labels);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((num_user_labels + dimBlock.x - 1) / dimBlock.x, (num_user_labels + dimBlock.y - 1) / dimBlock.y);
    invert_matrix_kernel<<<dimGrid, dimBlock>>>(d_G, d_P, num_user_labels);
    build_b_matrix_kernel<<<dimGrid, dimBlock>>>(d_B, d_P, num_user_labels);

    // Transpose matrix X
    transpose_kernel<<<dimGrid, dimBlock>>>(d_X, d_XT, num_user_labels, num_item_labels);

    // Multiply matrices X and XT
    matrix_multiply_kernel<<<dimGrid, dimBlock>>>(d_X, d_XT, d_C, num_user_labels, num_item_labels, num_user_labels);

    // Copy results back to host
    for (int i = 0; i < num_user_labels; i++) {
        hipMemcpy(P[i], &d_P[i * num_user_labels], num_user_labels * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(B[i], &d_B[i * num_user_labels], num_user_labels * sizeof(float), hipMemcpyDeviceToHost);
        if ((i + 1) % 5 == 0) {
            printf("Multiplication progress: Completed %d rows\n", i + 1);
        }
    }

    // Export B matrix to file
    export_matrix_to_file((float*)B, num_user_labels, num_user_labels, "output_matrix.txt");

    // Free device memory
    hipFree(d_G);
    hipFree(d_P);
    hipFree(d_B);
    hipFree(d_X);
    hipFree(d_XT);
    hipFree(d_C);
    hipFree(d_data);
    hipFree(d_user_labels);
    hipFree(d_item_labels);

    // Free host memory
    for (int i = 0; i < num_user_labels; i++) {
        free(G[i]);
        free(P[i]);
        free(B[i]);
    }
    free(G);
    free(P);
    free(B);
    free(data);
    free(user_labels);
    free(item_labels);

    auto end = std::chrono::high_resolution_clock::now(); // End time
    std::chrono::duration<double, std::milli> duration = end - start; // Calculate duration
    printf("Time elapsed: %f ms\n", duration.count()); // Print duration

    return 0;
}
